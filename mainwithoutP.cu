
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime_api.h"
#define FLDBL	double
const FLDBL PI=2.0*asin(1.);
const int BLOCK_X=16;
const int BLOCK_Y=16;
__global__ void streamingKernel(FLDBL *utemp,FLDBL *vtemp,FLDBL*un,FLDBL*vn,FLDBL*uup,FLDBL*udown,FLDBL*uleft,FLDBL*uright,FLDBL*vup,FLDBL*vdown,FLDBL*vleft,FLDBL*vright,FLDBL dx,FLDBL dy,FLDBL dt,int x, int y,FLDBL nu)
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
	
	int idx = tx+bx*blockDim.x;
	int jdx = ty+by*blockDim.y;
	if ((idx!=0)&&(idx!=x-1)&&(jdx!=0)&&(jdx!=y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==0)&&(jdx!=0)&&(jdx!=y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(uleft[jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+uleft[jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vleft[jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vleft[jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==x-1)&&(jdx!=0)&&(jdx!=y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-uright[jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-un[idx+x*(jdx+1)])/2/dy+nu*( (uright[jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vright[jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vright[jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx!=0)&&(idx!=x-1)&&(jdx==0))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(uup[idx]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+uup[idx])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vup[idx]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vup[idx])/dy/dy ));
	}
	else if ((idx!=0)&&(idx!=x-1)&&(jdx==y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-udown[idx])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (udown[idx]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vdown[idx])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vdown[idx]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==0)&&(jdx==0))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(uleft[jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(uup[idx]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+uleft[jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+uup[idx])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vleft[jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vup[idx]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vleft[jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vup[idx])/dy/dy ));
	}
	else if ((idx==0)&&(jdx==y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(uleft[jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-udown[idx])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+uleft[jdx])/dx/dx + (udown[idx]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vleft[jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vdown[idx])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vleft[jdx])/dx/dx + (vdown[idx]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==x-1)&&(jdx==0))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-uright[jdx])/2/dx+vn[idx+x*jdx]*(uup[idx]-un[idx+x*(jdx+1)])/2/dy+nu*( (uright[jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+uup[idx])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vright[jdx])/2/dx+vn[idx+x*jdx]*(vup[idx]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vright[jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vup[idx])/dy/dy ));
	}
	else if ((idx==x-1)&&(jdx==y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-uright[jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-udown[idx])/2/dy+nu*( (uright[jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (udown[idx]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vright[jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vdown[idx])/2/dy+nu*( (vright[jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vdown[idx]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
}
__global__ void lineKernel(FLDBL *utemp,FLDBL *vtemp,FLDBL*un,FLDBL*vn,FLDBL*uup,FLDBL*udown,FLDBL*uleft,FLDBL*uright,FLDBL*vup,FLDBL*vdown,FLDBL*vleft,FLDBL*vright,int x, int y)
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
	
	int idx = tx+bx*blockDim.x;
	int jdx = ty+by*blockDim.y;

	uup[idx]=utemp[idx+x*(y-1)];
	udown[idx]=utemp[idx];
	uleft[jdx]=utemp[x-1+x*jdx];
	uright[jdx]=utemp[x*jdx];
	vup[idx]=vtemp[idx+x*(y-1)];
	vdown[idx]=vtemp[idx];
	vleft[jdx]=vtemp[x-1+x*jdx];
	vright[jdx]=vtemp[x*jdx];
	un[idx+jdx*x]=utemp[idx+jdx*x];
	vn[idx+jdx*x]=vtemp[idx+jdx*x];
}
	//__shared__ FLDBL us[BLOCK_X][BLOCK_Y];
	//__shared__ FLDBL uns[BLOCK_X][BLOCK_Y];
	//__shared__ FLDBL vs[BLOCK_X][BLOCK_Y];
	//__shared__ FLDBL vns[BLOCK_X][BLOCK_Y];

	//for(i=1;i<x+1;i++)
	//	for(j=1;j<y+1;j++)
	//	{
	//		un[i+(x+2)*j]=u[i+(x+2)*j]+dt*(u[i+(x+2)*j]*(u[i-1+(x+2)*j]-u[i+1+(x+2)*j])/2/dx+v[i+(x+2)*j]*(u[i+(x+2)*(j-1)]-u[i+(x+2)*(j+1)])/2/dy+nu*( (u[i+1+(x+2)*j]-2*u[i+(x+2)*j]+u[i-2+(x+2)*j])/dx/dx + (u[i+(x+2)*(j+1)]-2*u[i+(x+2)*j]+u[i+(x+2)*(j-1)])/dy/dy ));
	//		vn[i+(x+2)*j]=v[i+(x+2)*j]+dt*(u[i+(x+2)*j]*(v[i-1+(x+2)*j]-v[i+1+(x+2)*j])/2/dx+v[i+(x+2)*j]*(v[i+(x+2)*(j-1)]-v[i+(x+2)*(j+1)])/2/dy+nu*( (v[i+1+(x+2)*j]-2*v[i+(x+2)*j]+v[i-2+(x+2)*j])/dx/dx + (v[i+(x+2)*(j+1)]-2*v[i+(x+2)*j]+v[i+(x+2)*(j-1)])/dy/dy ));
	//}
	//for(i=1;i<x+1;i++)
	//{
	//	un[i]=un[i+(x+2)*y];
	//	un[i+(x+2)*(y+1)]=un[i+x+2];
	//	vn[i]=vn[i+(x+2)*y];
	//	vn[i+(x+2)*(y+1)]=vn[i+x+2];
	//}
	//for(j=0;j<y+2;j++)
	//{
	//	un[(x+2)*j]=un[x+(x+2)*j];
	//	un[x+1+(x+2)*j]=un[1+(x+2)*j];
	//	vn[(x+2)*j]=vn[x+(x+2)*j];
	//	vn[x+1+(x+2)*j]=vn[1+(x+2)*j];
	//}
	////********FOURIE********


	////********RESULT********
	//for (i=1;i<x+1;i++)
	//	for(j=1;j<y+1;j++)
	//	{
	//		u[i+(x+2)*j]=un[i+(x+2)*j];
	//		v[i+(x+2)*j]=vn[i+(x+2)*j];
	//	}
	//for (int i=1;i<x+1;i++)
	//{
	//	u[i+(x+2)*0]=u[i+(x+2)*y];
	//	u[i+(x+2)*(y+1)]=u[i+(x+2)*1];
	//	v[i+(x+2)*0]=v[i+(x+2)*y];
	//	v[i+(x+2)*(y+1)]=v[i+(x+2)*1];
	//}
	//for (int j=0;j<y+2;j++)
	//{
	//	u[0+(x+2)*j]=u[x+(x+2)*j];
	//	u[x+1+(x+2)*j]=u[1+(x+2)*j];
	//	v[0+(x+2)*j]=v[x+(x+2)*j];
	//	v[x+1+(x+2)*j]=v[1+(x+2)*j];
	//}

	//us[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx];
	//vs[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx];

	//__syncthreads();
	//if ((tx>0)&&(tx<blockDim.x)&&(ty>0)&&(ty<blockDim.y))
	//{
	//	uns[tx][ty]=us[tx][ty]+dt*( us[tx][ty]*(us[tx-1][ty]-us[tx+1][ty])/2/dx+vs[tx][ty]*(us[tx][ty-1]-us[tx][ty+1])/2/dy+nu*( (us[tx+1][ty]-2.*us[tx][ty]+us[tx-1][ty])/dx/dx+(us[tx][ty+1]-2.*us[tx][ty]+us[tx][ty-1])/dy/dy ) );
	//	vns[tx][ty]=vs[tx][ty]+dt*( us[tx][ty]*(vs[tx-1][ty]-vs[tx+1][ty])/2/dx+vs[tx][ty]*(vs[tx][ty-1]-vs[tx][ty+1])/2/dy+nu*( (vs[tx+1][ty]-2.*vs[tx][ty]+vs[tx-1][ty])/dx/dx+(vs[tx][ty+1]-2.*vs[tx][ty]+vs[tx][ty-1])/dy/dy ) );
	//}
	//else if (!(((bx==0)&&(tx==0))||((bx==x/blockDim.x)&&(tx==blockDim.x))||((by==0)&&(ty==0))||((by==y/blockDim.y)&&(ty==blockDim.y))))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//else if ((bx==0)&&(tx==0))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(uleft[ty+by*blockDim.y]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+uleft[ty+by*blockDim.y])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vleft[ty+by*blockDim.y]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vleft[ty+by*blockDim.y])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//else if ((bx==x/blockDim.x)&&(tx==blockDim.x))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-uright[ty+by*blockDim.y])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (uright[ty+by*blockDim.y]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vright[ty+by*blockDim.y])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vright[ty+by*blockDim.y]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//else if ((by==0)&&(ty==0))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(uup[tx+bx*blockDim.x]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+uup[tx+bx*blockDim.x])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vup[tx+bx*blockDim.x]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vup[tx+bx*blockDim.x])/dy/dy ) );
	//}
	//else if ((by==y/blockDim.y)&&(ty==blockDim.y))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-udown[tx+bx*blockDim.x])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(udown[tx+bx*blockDim.x]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vdown[tx+bx*blockDim.x])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vdown[tx+bx*blockDim.x]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//__syncthreads();
	//un[tx+bx*blockDim.x+x*(ty+by*blockDim.y)]=uns[tx][ty];
	//vn[tx+bx*blockDim.x+x*(ty+by*blockDim.y)]=vns[tx][ty];
	//__syncthreads();
	//if ((by==y/blockDim.y)&&(ty==blockDim.y))
	//{
	//	uup[tx+bx*blockDim.x]=uns[tx][ty];
	//	vup[tx+bx*blockDim.x]=vns[tx][ty];
	//}
	//else if ((by==0)&&(ty==0))
	//{
	//	udown[tx+bx*blockDim.x]=uns[tx][ty];
	//	vdown[tx+bx*blockDim.x]=vns[tx][ty];
	//}
	//else if ((bx==x/blockDim.x)&&(tx==blockDim.x))
	//{
	//	uleft[ty+by*blockDim.y]=uns[tx][ty];
	//	vleft[ty+by*blockDim.y]=vns[tx][ty];
	//}	
	//else if ((bx==0)&&(tx==0))
	//{
	//	uright[ty+by*blockDim.y]=uns[tx][ty];
	//	vright[ty+by*blockDim.y]=vns[tx][ty];
	//}
	//__syncthreads();



void streaming (FLDBL*u,FLDBL*v,int x, int y,FLDBL dx, FLDBL dy, int t,FLDBL dt,  FLDBL nu );
void field_div(FLDBL * un, FLDBL * vn, int x, int y, FLDBL dx, FLDBL dy);
int main(int argc, char * argv[])
{
	FLDBL lx=2*PI;
	int x=256;
	int y=256;
	FLDBL dx=lx/x;
	FLDBL ly=2*PI;
	FLDBL dy=ly/y;
	int t=20;
	FLDBL dt=0.01;
	FLDBL lt=t*dt;
	FLDBL nu=0.1;

	FLDBL *u=new FLDBL[(x+2)*(y+2)];
	FLDBL *v=new FLDBL[(x+2)*(y+2)];

	for (int i=0;i<x+2;i++)
		for (int j=0;j<y+2;j++)
		{
			u[i+(x+2)*j]=-sin(2*PI*(j-1)/y);
			v[i+(x+2)*j]=sin(2*PI*(i-1)/x);
		};

	streaming(u,v,x,y,dx,dy,t,dt,nu);
	FILE *qout = fopen("q.dat", "w");
	for(int i = 0; i < x+2; i++)
	{
		for(int j = 0; j < y+2; j++)
		{
			if ((u[i+(x+2)*j]>1.)||(u[i+(x+2)*j]<-1.))
				u[i+(x+2)*j]=2.;
			fprintf(qout, "%lf\t%lf\t%lf\n", (double)i, (double)j, u[i+(x+2)*j]);
		}
		fprintf(qout, "\n");
	}
	fclose(qout);

	FILE *pout = fopen("p.dat", "w");
	for(int i = 0; i < x+2; i++)
	{
		for(int j = 0; j < y+2; j++)
		{
			if ((v[i+(x+2)*j]>1.)||(v[i+(x+2)*j]<-1.))
				v[i+(x+2)*j]=2.;
			fprintf(pout, "%lf\t%lf\t%lf\n", (double)i, (double)j, v[i+(x+2)*j]);
		}
		fprintf(pout, "\n");
	}
	fclose(pout);

	delete [] u;
	delete [] v;

	return 0;
}
void streaming (FLDBL*u,FLDBL*v,int x, int y,FLDBL dx, FLDBL dy, int t,FLDBL dt, FLDBL nu )
{
	FLDBL * uup=new FLDBL[x];
	FLDBL * udown=new FLDBL[x];
	FLDBL * uleft=new FLDBL[y];
	FLDBL * uright=new FLDBL[y];
	FLDBL * vup=new FLDBL[x];
	FLDBL * vdown=new FLDBL[x];
	FLDBL * vleft=new FLDBL[y];
	FLDBL * vright=new FLDBL[y];
	FLDBL * un=new FLDBL[x*y];
	FLDBL * vn=new FLDBL[x*y];
	int i=0;
	int j=0;
	for (i=1;i<x+1;i++)
		for(j=1;j<y+1;j++)
		{
			un[i-1+x*(j-1)]=u[i+(x+2)*j];
			vn[i-1+x*(j-1)]=v[i+(x+2)*j];
		}
	//FILE *pout = fopen("p.dat", "w");
	//for(int i = 0; i < x; i++)
	//{
	//	for(int j = 0; j < y; j++)
	//	{
	//		fprintf(pout, "%lf\t%lf\t%lf\n", (double)i, (double)j, vn[i+x*j]);
	//	}
	//	fprintf(pout, "\n");
	//}
	//fclose(pout);

	for(i=1;i<x+1;i++)
	{
		uup[i-1]=u[i];
		udown[i-1]=u[i+(x+2)*(y+1)];
		vup[i-1]=v[i];
		vdown[i-1]=v[i+(x+2)*(y+1)];
	}
	for(j=1;j<y+1;j++)
	{
		uleft[j-1]=u[(x+2)*j];
		uright[j-1]=u[x+1+(x+2)*j];
		vleft[j-1]=v[(x+2)*j];
		vright[j-1]=v[x+1+(x+2)*j];
	}

	int numBytes = x*y*sizeof(FLDBL);
	FLDBL*unDev=NULL;
	FLDBL*vnDev=NULL;
	FLDBL*uleftDev=NULL;
	FLDBL*urightDev=NULL;
	FLDBL*uupDev=NULL;
	FLDBL*udownDev=NULL;
	FLDBL*vleftDev=NULL;
	FLDBL*vrightDev=NULL;
	FLDBL*vupDev=NULL;
	FLDBL*vdownDev=NULL;
	FLDBL*utempDev=NULL;
	FLDBL*vtempDev=NULL;
	dim3 threads = dim3(BLOCK_X,BLOCK_Y);
	dim3 blocks = dim3(x/threads.x,y/threads.y);
	hipMalloc((void**)& unDev,numBytes);
	hipMalloc((void**)& vnDev,numBytes);
	hipMalloc((void**)& uupDev,x*sizeof(FLDBL));
	hipMalloc((void**)& udownDev,x*sizeof(FLDBL));
	hipMalloc((void**)& uleftDev,y*sizeof(FLDBL));
	hipMalloc((void**)& urightDev,y*sizeof(FLDBL));
	hipMalloc((void**)& vupDev,x*sizeof(FLDBL));
	hipMalloc((void**)& vdownDev,x*sizeof(FLDBL));
	hipMalloc((void**)& vleftDev,y*sizeof(FLDBL));
	hipMalloc((void**)& vrightDev,y*sizeof(FLDBL));
	hipMalloc((void**)& utempDev,numBytes);
	hipMalloc((void**)& vtempDev,numBytes);
	hipMemcpy(unDev,un,numBytes, hipMemcpyHostToDevice);
	hipMemcpy(vnDev,vn,numBytes, hipMemcpyHostToDevice);
	hipMemcpy(uupDev,uup,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(udownDev,udown,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(uleftDev,uleft,y*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(urightDev,uright,y*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vupDev,vup,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vdownDev,vdown,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vleftDev,vleft,y*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vrightDev,vright,y*sizeof(FLDBL),hipMemcpyHostToDevice);
	for (int ti=0;ti<t;ti++)
	{
		if (!(ti%100))
			printf("%d from %d\n",ti/100,t/100);

		streamingKernel<<<blocks,threads>>>(utempDev,vtempDev,unDev,vnDev,uupDev,udownDev,uleftDev, urightDev,vupDev,vdownDev,vleftDev, vrightDev,dx,dy,dt,x,y,nu);
		lineKernel<<<blocks,threads>>>(utempDev,vtempDev,unDev,vnDev,uupDev,udownDev,uleftDev, urightDev,vupDev,vdownDev,vleftDev, vrightDev,x,y);
		printf("%s\n",hipGetErrorString(hipGetLastError()));
	}
	hipMemcpy(un,unDev,numBytes,hipMemcpyDeviceToHost);
	hipMemcpy(vn,vnDev,numBytes,hipMemcpyDeviceToHost);
	hipMemcpy(uup,uupDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(udown,udownDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(uleft,uleftDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(uright,urightDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vup,vupDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vdown,vdownDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vleft,vleftDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vright,vrightDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipFree(unDev);
	hipFree(vnDev);
	hipFree(uupDev);
	hipFree(udownDev);
	hipFree(uleftDev);
	hipFree(urightDev);
	hipFree(vupDev);
	hipFree(vdownDev);
	hipFree(vleftDev);
	hipFree(vrightDev);
	hipFree(utempDev);
	hipFree(vtempDev);
	field_div(un,vn,x,y,dx,dy);
	for (i=1;i<x+1;i++)
		for(j=1;j<y+1;j++)
		{
			u[i+(x+2)*j]=un[i-1+x*(j-1)];
			v[i+(x+2)*j]=vn[i-1+x*(j-1)];
		}
	for(i=1;i<x+1;i++)
	{
		u[i]=uup[i-1];
		u[i+(x+2)*(y+1)]=udown[i-1];
		v[i]=vup[i-1];
		v[i+(x+2)*(y+1)]=vdown[i-1];
	}
	for(j=1;j<y+1;j++)
	{
		u[(x+2)*j]=uleft[j-1];
		u[x+1+(x+2)*j]=uright[j-1];
		v[(x+2)*j]=vleft[j-1];
		v[x+1+(x+2)*j]=vright[j-1];
	}
	delete [] un;
	delete [] vn;
	delete [] uleft;
	delete [] uright;
	delete [] uup;
	delete [] udown;
}
void field_div(FLDBL * un, FLDBL * vn, int x, int y, FLDBL dx, FLDBL dy)
{
	FLDBL divu=0;
	int maxi=0;
	int maxj=0;
	int i=0;
	int j=0;
	divu=0;
	for (i=1;i<x-1;i++)
		for (j=1;j<y-1;j++)
			if(divu<fabs((un[i+1+x*j]-un[i-1+x*j])/dx/2+(vn[i+x*(j+1)]-vn[i+x*(j-1)])/dy/2))
			{
				divu=fabs((un[i+1+x*j]-un[i-1+x*j])/dx/2+(vn[i+x*(j+1)]-vn[i+x*(j-1)])/dy/2);
				maxi=i;
				maxj=j;
			}
	printf("~~~i=%d,j=%d~\n",maxi,maxj);
	printf("~~~~div = %f;\n",divu);
	FLDBL omega=0;
	for (i=0;i<x-1;i++)
		for (j=0;j<y-1;j++)
			if(omega<abs(-(un[i+x*(j+1)]-un[i+x*j])/dy+(vn[i+1+x*j]-vn[i+x*j])/dx))
				omega=abs(-(un[i+x*(j+1)]-un[i+x*j])/dy+(vn[i+1+x*j]-vn[i+x*j])/dx);
	printf("~~~~omega=%f\n",omega);

}