#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
 
#include <hipfft/hipfft.h>

#define FLDBL	double
const FLDBL PI=2.0*asin(1.);
const int BLOCK_X=16;
const int BLOCK_Y=16;
__global__ void streamingKernel(FLDBL *utemp,FLDBL *vtemp,FLDBL*un,FLDBL*vn,FLDBL*uup,FLDBL*udown,FLDBL*uleft,FLDBL*uright,FLDBL*vup,FLDBL*vdown,FLDBL*vleft,FLDBL*vright,FLDBL dx,FLDBL dy,FLDBL dt,int x, int y,FLDBL nu)
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
	
	int idx = tx+bx*blockDim.x;
	int jdx = ty+by*blockDim.y;
	if ((idx!=0)&&(idx!=x-1)&&(jdx!=0)&&(jdx!=y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==0)&&(jdx!=0)&&(jdx!=y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(uleft[jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+uleft[jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vleft[jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vleft[jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==x-1)&&(jdx!=0)&&(jdx!=y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-uright[jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-un[idx+x*(jdx+1)])/2/dy+nu*( (uright[jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vright[jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vright[jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx!=0)&&(idx!=x-1)&&(jdx==0))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(uup[idx]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+uup[idx])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vup[idx]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vup[idx])/dy/dy ));
	}
	else if ((idx!=0)&&(idx!=x-1)&&(jdx==y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-udown[idx])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (udown[idx]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vdown[idx])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vdown[idx]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==0)&&(jdx==0))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(uleft[jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(uup[idx]-un[idx+x*(jdx+1)])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+uleft[jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+uup[idx])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vleft[jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vup[idx]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vleft[jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vup[idx])/dy/dy ));
	}
	else if ((idx==0)&&(jdx==y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(uleft[jdx]-un[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-udown[idx])/2/dy+nu*( (un[idx+1+x*jdx]-2*un[idx+x*jdx]+uleft[jdx])/dx/dx + (udown[idx]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vleft[jdx]-vn[idx+1+x*jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vdown[idx])/2/dy+nu*( (vn[idx+1+x*jdx]-2*vn[idx+x*jdx]+vleft[jdx])/dx/dx + (vdown[idx]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
	else if ((idx==x-1)&&(jdx==0))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-uright[jdx])/2/dx+vn[idx+x*jdx]*(uup[idx]-un[idx+x*(jdx+1)])/2/dy+nu*( (uright[jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (un[idx+x*(jdx+1)]-2*un[idx+x*jdx]+uup[idx])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vright[jdx])/2/dx+vn[idx+x*jdx]*(vup[idx]-vn[idx+x*(jdx+1)])/2/dy+nu*( (vright[jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vn[idx+x*(jdx+1)]-2*vn[idx+x*jdx]+vup[idx])/dy/dy ));
	}
	else if ((idx==x-1)&&(jdx==y-1))
	{
		utemp[idx+x*jdx]=un[idx+x*jdx]+dt*(un[idx+x*jdx]*(un[idx-1+x*jdx]-uright[jdx])/2/dx+vn[idx+x*jdx]*(un[idx+x*(jdx-1)]-udown[idx])/2/dy+nu*( (uright[jdx]-2*un[idx+x*jdx]+un[idx-1+x*jdx])/dx/dx + (udown[idx]-2*un[idx+x*jdx]+un[idx+x*(jdx-1)])/dy/dy ));
		vtemp[idx+x*jdx]=vn[idx+x*jdx]+dt*(un[idx+x*jdx]*(vn[idx-1+x*jdx]-vright[jdx])/2/dx+vn[idx+x*jdx]*(vn[idx+x*(jdx-1)]-vdown[idx])/2/dy+nu*( (vright[jdx]-2*vn[idx+x*jdx]+vn[idx-1+x*jdx])/dx/dx + (vdown[idx]-2*vn[idx+x*jdx]+vn[idx+x*(jdx-1)])/dy/dy ));
	}
}
__global__ void lineKernel(FLDBL *utemp,FLDBL *vtemp,FLDBL *f,int x, int y,FLDBL dx,FLDBL dy)
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
	
	int idx = tx+bx*blockDim.x;
	int jdx = ty+by*blockDim.y;

	if ((idx!=0)&&(idx!=x-1)&&(jdx!=0)&&(jdx!=y-1))
		f[jdx+idx*y]=(utemp[idx+1+jdx*x]-utemp[idx-1+jdx*x])/2/dx+(vtemp[idx+x*(jdx+1)]-vtemp[idx+x*(jdx-1)])/2/dy;
	else if ((idx==0)&&(jdx!=0)&&(jdx!=y-1))
		f[jdx+idx*y]=(utemp[idx+1+jdx*x]-utemp[x-1+x*jdx])/2/dx+(vtemp[idx+x*(jdx+1)]-vtemp[idx+x*(jdx-1)])/2/dy;
	else if ((idx==x-1)&&(jdx!=0)&&(jdx!=y-1))
		f[jdx+idx*y]=(utemp[x*jdx]-utemp[idx-1+jdx*x])/2/dx+(vtemp[idx+x*(jdx+1)]-vtemp[idx+x*(jdx-1)])/2/dy;
	else if ((idx!=0)&&(idx!=x-1)&&(jdx==0))
		f[jdx+idx*y]=(utemp[idx+1+jdx*x]-utemp[idx-1+jdx*x])/2/dx+(vtemp[idx+x*(jdx+1)]-vtemp[idx+x*(y-1)])/2/dy;
	else if ((idx!=0)&&(idx!=x-1)&&(jdx==y-1))
		f[jdx+idx*y]=(utemp[idx+1+jdx*x]-utemp[idx-1+jdx*x])/2/dx+(vtemp[idx]-vtemp[idx+x*(jdx-1)])/2/dy;
	else if ((idx==0)&&(jdx==0))
		f[jdx+idx*y]=(utemp[idx+1+jdx*x]-utemp[x-1+x*jdx])/2/dx+(vtemp[idx+x*(jdx+1)]-vtemp[idx+x*(y-1)])/2/dy;
	else if ((idx==0)&&(jdx==y-1))
		f[jdx+idx*y]=(utemp[idx+1+jdx*x]-utemp[x-1+x*jdx])/2/dx+(vtemp[idx]-vtemp[idx+x*(jdx-1)])/2/dy;
	else if ((idx==x-1)&&(jdx==0))
		f[jdx+idx*y]=(utemp[x*jdx]-utemp[idx-1+jdx*x])/2/dx+(vtemp[idx+x*(jdx+1)]-vtemp[idx+x*(y-1)])/2/dy;
	else if ((idx==x-1)&&(jdx==y-1))
		f[jdx+idx*y]=(utemp[x*jdx]-utemp[idx-1+jdx*x])/2/dx+(vtemp[idx]-vtemp[idx+x*(jdx-1)])/2/dy;
}
__global__ void fourieKernel(hipfftDoubleComplex * data, FLDBL dx, FLDBL dy, int x, int y)
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
	
	int idx = tx+bx*blockDim.x+1;
	int jdx = ty+by*blockDim.y+1;

	data[jdx+(y/2+1)*idx].x=data[jdx+(y/2+1)*idx].x/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx-4*sin(dy*jdx/2)*sin(dy*jdx/2)/dy/dy);
	if (idx!=x/2)
		data[jdx+(y/2+1)*(x-idx)].x=data[jdx+(y/2+1)*(x-idx)].x/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx-4*sin(dy*jdx/2)*sin(dy*jdx/2)/dy/dy);
	data[(y/2+1)*idx].x=data[(y/2+1)*idx].x/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx);
	if (idx!=x/2)
		data[(y/2+1)*(x-idx)].x=data[(y/2+1)*(x-idx)].x/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx);
	data[jdx].x=data[jdx].x/(-4*sin(dy*jdx/2)*sin(dy*jdx/2)/dy/dy);
	data[jdx+(y/2+1)*idx].y=data[jdx+(y/2+1)*idx].y/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx-4*sin(dy*jdx/2)*sin(dy*jdx/2)/dy/dy);
	if (idx!=x/2)
		data[jdx+(y/2+1)*(x-idx)].y=data[jdx+(y/2+1)*(x-idx)].y/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx-4*sin(dy*jdx/2)*sin(dy*jdx/2)/dy/dy);
	data[(y/2+1)*idx].y=data[(y/2+1)*idx].y/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx);
	if (idx!=x/2)
		data[(y/2+1)*(x-idx)].y=data[(y/2+1)*(x-idx)].y/(-4*sin(dx*idx/2)*sin(dx*idx/2)/dx/dx);
	data[jdx].y=data[jdx].y/(-4*sin(dy*jdx/2)*sin(dy*jdx/2)/dy/dy);
}
__global__ void summaryKernel(FLDBL * p,FLDBL *utemp,FLDBL *vtemp,FLDBL*un,FLDBL*vn,FLDBL*uup,FLDBL*udown,FLDBL*uleft,FLDBL*uright,FLDBL*vup,FLDBL*vdown,FLDBL*vleft,FLDBL*vright,int x, int y, FLDBL dx, FLDBL dy)
{
	int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
	
	int idx = tx+bx*blockDim.x+1;
	int jdx = ty+by*blockDim.y+1;
	if ((idx<x-1)&&(jdx<y-1))
	{
	un[idx+jdx*x]=utemp[idx+jdx*x]-(p[idx+1+x*jdx]-p[idx-1+x*jdx])/2/dx;

	un[idx]=utemp[idx]-(p[idx+1]-p[idx-1])/2/dx;
	un[idx+x*(y-1)]=utemp[idx+x*(y-1)]-(p[idx+1+x*(y-1)]-p[idx-1+x*(y-1)])/2/dx;
	un[jdx*x]=utemp[jdx*x]-(p[1+x*jdx]-p[x-1+x*jdx])/2/dx;
	un[x-1+jdx*x]=utemp[x-1+jdx*x]-(p[x*jdx]-p[x-2+x*jdx])/2/dx;

	vn[idx+jdx*x]=vtemp[idx+jdx*x]-(p[idx+x*(jdx+1)]-p[idx+x*(jdx-1)])/2/dy;

	vn[idx]=vtemp[idx]-(p[idx+x]-p[idx+x*(y-1)])/2/dy;
	vn[idx+x*(y-1)]=vtemp[idx+x*(y-1)]-(p[idx]-p[idx+x*(y-2)])/dy/2;
	vn[jdx*x]=vtemp[jdx*x]-(p[x*(jdx+1)]-p[x*(jdx-1)])/2/dy;
	vn[x-1+jdx*x]=vtemp[x-1+jdx*x]-(p[x-1+x*(jdx+1)]-p[x-1+x*(jdx-1)])/2/dy;

	un[0+x*0]=utemp[0+x*0]-(p[1]-p[x-1])/dx/2;
	un[x-1+x*0]=utemp[x-1+x*0]-(p[0]-p[x-2])/dx/2;
	un[0+x*(y-1)]=utemp[0+x*(y-1)]-(p[1+x*(y-1)]-p[x-1+x*(y-1)])/dx/2;
	un[x-1+x*(y-1)]=utemp[x-1+x*(y-1)]-(p[x*(y-1)]-p[x-2+x*(y-1)])/dx/2;

	vn[0+x*0]=vtemp[0+x*0]-(p[x]-p[x*(y-1)])/dy/2;
	vn[x-1+x*0]=vtemp[x-1+x*0]-(p[x-1+x]-p[x-1+x*(y-1)])/dy/2;
	vn[0+x*(y-1)]=vtemp[0+x*(y-1)]-(p[0]-p[x*(y-2)])/dy/2;
	vn[x-1+x*(y-1)]=vtemp[x-1+x*(y-1)]-(p[x-1+0]-p[x-1+x*(y-2)])/dy/2;

	uup[idx]=utemp[idx+x*(y-1)]-(p[idx+1+x*(y-1)]-p[idx-1+x*(y-1)])/2/dx;
	udown[idx]=utemp[idx]-(p[idx+1]-p[idx-1])/2/dx;
	uleft[jdx]=utemp[x-1+jdx*x]-(p[x*jdx]-p[x-2+x*jdx])/2/dx;
	uright[jdx]=utemp[jdx*x]-(p[1+x*jdx]-p[x-1+x*jdx])/2/dx;

	vup[idx]=vtemp[idx+x*(y-1)]-(p[idx]-p[idx+x*(y-2)])/dy/2;
	vdown[idx]=vtemp[idx]-(p[idx+x]-p[idx+x*(y-1)])/2/dy;
	vleft[jdx]=vtemp[x-1+jdx*x]-(p[x-1+x*(jdx+1)]-p[x-1+x*(jdx-1)])/2/dy;
	vright[jdx]=vtemp[jdx*x]-(p[x*(jdx+1)]-p[x*(jdx-1)])/2/dy;

	udown[0]=utemp[0+x*0]-(p[1]-p[x-1])/dx/2;
	uright[0]=utemp[0+x*0]-(p[1]-p[x-1])/dx/2;
	uleft[0]=utemp[x-1+x*0]-(p[0]-p[x-2])/dx/2;
	udown[x-1]=utemp[x-1+x*0]-(p[0]-p[x-2])/dx/2;
	uup[0]=utemp[0+x*(y-1)]-(p[1+x*(y-1)]-p[x-1+x*(y-1)])/dx/2;
	uright[y-1]=utemp[0+x*(y-1)]-(p[1+x*(y-1)]-p[x-1+x*(y-1)])/dx/2;
	uup[x-1]=utemp[x-1+x*(y-1)]-(p[x*(y-1)]-p[x-2+x*(y-1)])/dx/2;
	uleft[y-1]=utemp[x-1+x*(y-1)]-(p[x*(y-1)]-p[x-2+x*(y-1)])/dx/2;

	vright[0]=vtemp[0+x*0]-(p[x]-p[x*(y-1)])/dy/2;
	vdown[0]=vtemp[0+x*0]-(p[x]-p[x*(y-1)])/dy/2;
	vleft[0]=vtemp[x-1+x*0]-(p[x-1+x]-p[x-1+x*(y-1)])/dy/2;
	vdown[x-1]=vtemp[x-1+x*0]-(p[x-1+x]-p[x-1+x*(y-1)])/dy/2;
	vright[y-1]=vtemp[0+x*(y-1)]-(p[0]-p[x*(y-2)])/dy/2;
	vup[0]=vtemp[0+x*(y-1)]-(p[0]-p[x*(y-2)])/dy/2;
	vup[y-1]=vtemp[x-1+x*(y-1)]-(p[x-1+0]-p[x-1+x*(y-2)])/dy/2;
	vleft[x-1]=vtemp[x-1+x*(y-1)]-(p[x-1+0]-p[x-1+x*(y-2)])/dy/2;
	}
}


	//__shared__ FLDBL us[BLOCK_X][BLOCK_Y];
	//__shared__ FLDBL uns[BLOCK_X][BLOCK_Y];
	//__shared__ FLDBL vs[BLOCK_X][BLOCK_Y];
	//__shared__ FLDBL vns[BLOCK_X][BLOCK_Y];

	//for(i=1;i<x+1;i++)
	//	for(j=1;j<y+1;j++)
	//	{
	//		un[i+(x+2)*j]=u[i+(x+2)*j]+dt*(u[i+(x+2)*j]*(u[i-1+(x+2)*j]-u[i+1+(x+2)*j])/2/dx+v[i+(x+2)*j]*(u[i+(x+2)*(j-1)]-u[i+(x+2)*(j+1)])/2/dy+nu*( (u[i+1+(x+2)*j]-2*u[i+(x+2)*j]+u[i-2+(x+2)*j])/dx/dx + (u[i+(x+2)*(j+1)]-2*u[i+(x+2)*j]+u[i+(x+2)*(j-1)])/dy/dy ));
	//		vn[i+(x+2)*j]=v[i+(x+2)*j]+dt*(u[i+(x+2)*j]*(v[i-1+(x+2)*j]-v[i+1+(x+2)*j])/2/dx+v[i+(x+2)*j]*(v[i+(x+2)*(j-1)]-v[i+(x+2)*(j+1)])/2/dy+nu*( (v[i+1+(x+2)*j]-2*v[i+(x+2)*j]+v[i-2+(x+2)*j])/dx/dx + (v[i+(x+2)*(j+1)]-2*v[i+(x+2)*j]+v[i+(x+2)*(j-1)])/dy/dy ));
	//}
	//for(i=1;i<x+1;i++)
	//{
	//	un[i]=un[i+(x+2)*y];
	//	un[i+(x+2)*(y+1)]=un[i+x+2];
	//	vn[i]=vn[i+(x+2)*y];
	//	vn[i+(x+2)*(y+1)]=vn[i+x+2];
	//}
	//for(j=0;j<y+2;j++)
	//{
	//	un[(x+2)*j]=un[x+(x+2)*j];
	//	un[x+1+(x+2)*j]=un[1+(x+2)*j];
	//	vn[(x+2)*j]=vn[x+(x+2)*j];
	//	vn[x+1+(x+2)*j]=vn[1+(x+2)*j];
	//}
	////********FOURIE********


	////********RESULT********
	//for (i=1;i<x+1;i++)
	//	for(j=1;j<y+1;j++)
	//	{
	//		u[i+(x+2)*j]=un[i+(x+2)*j];
	//		v[i+(x+2)*j]=vn[i+(x+2)*j];
	//	}
	//for (int i=1;i<x+1;i++)
	//{
	//	u[i+(x+2)*0]=u[i+(x+2)*y];
	//	u[i+(x+2)*(y+1)]=u[i+(x+2)*1];
	//	v[i+(x+2)*0]=v[i+(x+2)*y];
	//	v[i+(x+2)*(y+1)]=v[i+(x+2)*1];
	//}
	//for (int j=0;j<y+2;j++)
	//{
	//	u[0+(x+2)*j]=u[x+(x+2)*j];
	//	u[x+1+(x+2)*j]=u[1+(x+2)*j];
	//	v[0+(x+2)*j]=v[x+(x+2)*j];
	//	v[x+1+(x+2)*j]=v[1+(x+2)*j];
	//}

	//us[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx];
	//vs[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx];

	//__syncthreads();
	//if ((tx>0)&&(tx<blockDim.x)&&(ty>0)&&(ty<blockDim.y))
	//{
	//	uns[tx][ty]=us[tx][ty]+dt*( us[tx][ty]*(us[tx-1][ty]-us[tx+1][ty])/2/dx+vs[tx][ty]*(us[tx][ty-1]-us[tx][ty+1])/2/dy+nu*( (us[tx+1][ty]-2.*us[tx][ty]+us[tx-1][ty])/dx/dx+(us[tx][ty+1]-2.*us[tx][ty]+us[tx][ty-1])/dy/dy ) );
	//	vns[tx][ty]=vs[tx][ty]+dt*( us[tx][ty]*(vs[tx-1][ty]-vs[tx+1][ty])/2/dx+vs[tx][ty]*(vs[tx][ty-1]-vs[tx][ty+1])/2/dy+nu*( (vs[tx+1][ty]-2.*vs[tx][ty]+vs[tx-1][ty])/dx/dx+(vs[tx][ty+1]-2.*vs[tx][ty]+vs[tx][ty-1])/dy/dy ) );
	//}
	//else if (!(((bx==0)&&(tx==0))||((bx==x/blockDim.x)&&(tx==blockDim.x))||((by==0)&&(ty==0))||((by==y/blockDim.y)&&(ty==blockDim.y))))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//else if ((bx==0)&&(tx==0))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(uleft[ty+by*blockDim.y]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+uleft[ty+by*blockDim.y])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vleft[ty+by*blockDim.y]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vleft[ty+by*blockDim.y])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//else if ((bx==x/blockDim.x)&&(tx==blockDim.x))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-uright[ty+by*blockDim.y])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (uright[ty+by*blockDim.y]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vright[ty+by*blockDim.y])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vright[ty+by*blockDim.y]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//else if ((by==0)&&(ty==0))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(uup[tx+bx*blockDim.x]-un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(un[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+uup[tx+bx*blockDim.x])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vup[tx+bx*blockDim.x]-vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vn[(ty+by*blockDim.y+1)*x+tx+blockDim.x*bx]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vup[tx+bx*blockDim.x])/dy/dy ) );
	//}
	//else if ((by==y/blockDim.y)&&(ty==blockDim.y))
	//{
	//	uns[tx][ty]=un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-udown[tx+bx*blockDim.x])/2/dy+nu*( (un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(udown[tx+bx*blockDim.x]-2.*un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+un[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//	vns[tx][ty]=vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+dt*( un[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1]-vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1])/2/dx+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]*(vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx]-vdown[tx+bx*blockDim.x])/2/dy+nu*( (vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx+1]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx-1])/dx/dx+(vdown[tx+bx*blockDim.x]-2.*vn[(ty+by*blockDim.y)*x+tx+blockDim.x*bx]+vn[(ty+by*blockDim.y-1)*x+tx+blockDim.x*bx])/dy/dy ) );
	//}
	//__syncthreads();
	//un[tx+bx*blockDim.x+x*(ty+by*blockDim.y)]=uns[tx][ty];
	//vn[tx+bx*blockDim.x+x*(ty+by*blockDim.y)]=vns[tx][ty];
	//__syncthreads();
	//if ((by==y/blockDim.y)&&(ty==blockDim.y))
	//{
	//	uup[tx+bx*blockDim.x]=uns[tx][ty];
	//	vup[tx+bx*blockDim.x]=vns[tx][ty];
	//}
	//else if ((by==0)&&(ty==0))
	//{
	//	udown[tx+bx*blockDim.x]=uns[tx][ty];
	//	vdown[tx+bx*blockDim.x]=vns[tx][ty];
	//}
	//else if ((bx==x/blockDim.x)&&(tx==blockDim.x))
	//{
	//	uleft[ty+by*blockDim.y]=uns[tx][ty];
	//	vleft[ty+by*blockDim.y]=vns[tx][ty];
	//}	
	//else if ((bx==0)&&(tx==0))
	//{
	//	uright[ty+by*blockDim.y]=uns[tx][ty];
	//	vright[ty+by*blockDim.y]=vns[tx][ty];
	//}
	//__syncthreads();



void streaming (FLDBL*u,FLDBL*v,int x, int y,FLDBL dx, FLDBL dy, int t,FLDBL dt,  FLDBL nu );
void field_div(FLDBL * un, FLDBL * vn, int x, int y, FLDBL dx, FLDBL dy);
int main(int argc, char * argv[])
{
	FLDBL lx=2*PI;
	int x=256;
	int y=256;
	FLDBL dx=lx/x;
	FLDBL ly=2*PI;
	FLDBL dy=ly/y;
	int t=100;
	FLDBL dt=0.001;
	//FLDBL lt=t*dt;
	FLDBL nu=0.001;

	FLDBL *u=new FLDBL[(x+2)*(y+2)];
	FLDBL *v=new FLDBL[(x+2)*(y+2)];

	for (int i=0;i<x+2;i++)
		for (int j=0;j<y+2;j++)
		{
			u[i+(x+2)*j]=-sin(2*PI*(j-1)/y);
			v[i+(x+2)*j]=sin(2*PI*(i-1)/x);
		};

	streaming(u,v,x,y,dx,dy,t,dt,nu);
	FILE *qout = fopen("q.dat", "w");
	for(int i = 1; i < x+1; i++)
	{
		for(int j = 1; j < y+1; j++)
		{
			if ((u[i+(x+2)*j]>2.)||(u[i+(x+2)*j]<-2.))
				u[i+(x+2)*j]=3.;
			fprintf(qout, "%lf\t%lf\t%lf\n", 2*PI*(i-1)/x, 2*PI*(j-1)/y, -(u[i+(x+2)*(j+1)]-u[i+(x+2)*(j-1)])/2/dy+(v[i+1+(x+2)*j]-v[i-1+(x+2)*j])/2/dx);
		}
		fprintf(qout, "\n");
	}
	fclose(qout);

	FILE *pout = fopen("p.dat", "w");
	for(int i = 0; i < x+2; i++)
	{
		for(int j = 0; j < y+2; j++)
		{
			if ((v[i+(x+2)*j]>2.)||(v[i+(x+2)*j]<-2.))
				v[i+(x+2)*j]=3.;
			fprintf(pout, "%lf\t%lf\t%lf\n", (double)i, (double)j, u[i+(x+2)*j]+sin(2*PI*(j-1)/y)*exp(-nu*dt*t));
		}
		fprintf(pout, "\n");
	}
	fclose(pout);

	delete [] u;
	delete [] v;

	return 0;
}
void streaming (FLDBL*u,FLDBL*v,int x, int y,FLDBL dx, FLDBL dy, int t,FLDBL dt, FLDBL nu )
{
	FLDBL * uup=new FLDBL[x];
	FLDBL * udown=new FLDBL[x];
	FLDBL * uleft=new FLDBL[y];
	FLDBL * uright=new FLDBL[y];
	FLDBL * vup=new FLDBL[x];
	FLDBL * vdown=new FLDBL[x];
	FLDBL * vleft=new FLDBL[y];
	FLDBL * vright=new FLDBL[y];
	FLDBL * un=new FLDBL[x*y];
	FLDBL * vn=new FLDBL[x*y];
	FLDBL * f=new FLDBL[x*y];
	FLDBL * p=new FLDBL[x*y];

	int i=0;
	int j=0;
	for (i=1;i<x+1;i++)
		for(j=1;j<y+1;j++)
		{
			un[i-1+x*(j-1)]=u[i+(x+2)*j];
			vn[i-1+x*(j-1)]=v[i+(x+2)*j];
		}
	//FILE *pout = fopen("p.dat", "w");
	//for(int i = 0; i < x; i++)
	//{
	//	for(int j = 0; j < y; j++)
	//	{
	//		fprintf(pout, "%lf\t%lf\t%lf\n", (double)i, (double)j, vn[i+x*j]);
	//	}
	//	fprintf(pout, "\n");
	//}
	//fclose(pout);

	for(i=1;i<x+1;i++)
	{
		uup[i-1]=u[i];
		udown[i-1]=u[i+(x+2)*(y+1)];
		vup[i-1]=v[i];
		vdown[i-1]=v[i+(x+2)*(y+1)];
	}
	for(j=1;j<y+1;j++)
	{
		uleft[j-1]=u[(x+2)*j];
		uright[j-1]=u[x+1+(x+2)*j];
		vleft[j-1]=v[(x+2)*j];
		vright[j-1]=v[x+1+(x+2)*j];
	}

	int numBytes = x*y*sizeof(FLDBL);
	FLDBL*unDev=NULL;
	FLDBL*vnDev=NULL;
	FLDBL*uleftDev=NULL;
	FLDBL*urightDev=NULL;
	FLDBL*uupDev=NULL;
	FLDBL*udownDev=NULL;
	FLDBL*vleftDev=NULL;
	FLDBL*vrightDev=NULL;
	FLDBL*vupDev=NULL;
	FLDBL*vdownDev=NULL;
	FLDBL*utempDev=NULL;
	FLDBL*vtempDev=NULL;
	FLDBL*fDev=NULL;
	FLDBL*pDev=NULL;
	dim3 threads = dim3(BLOCK_X,BLOCK_Y);
	dim3 blocks = dim3(x/threads.x,y/threads.y);
	dim3 sblocks = dim3(x/2/threads.x,y/2/threads.y);
	dim3 lsblocks = dim3((x-2)/threads.x+1, (y-2)/threads.y+1);
	hipMalloc((void**)& unDev,numBytes);
	hipMalloc((void**)& vnDev,numBytes);
	hipMalloc((void**)& uupDev,x*sizeof(FLDBL));
	hipMalloc((void**)& udownDev,x*sizeof(FLDBL));
	hipMalloc((void**)& uleftDev,y*sizeof(FLDBL));
	hipMalloc((void**)& urightDev,y*sizeof(FLDBL));
	hipMalloc((void**)& vupDev,x*sizeof(FLDBL));
	hipMalloc((void**)& vdownDev,x*sizeof(FLDBL));
	hipMalloc((void**)& vleftDev,y*sizeof(FLDBL));
	hipMalloc((void**)& vrightDev,y*sizeof(FLDBL));
	hipMalloc((void**)& utempDev,numBytes);
	hipMalloc((void**)& vtempDev,numBytes);
	hipMalloc((void**)& fDev,numBytes);
	hipMalloc((void**)& pDev,numBytes);
	hipMemcpy(unDev,un,numBytes, hipMemcpyHostToDevice);
	hipMemcpy(vnDev,vn,numBytes, hipMemcpyHostToDevice);
	hipMemcpy(uupDev,uup,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(udownDev,udown,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(uleftDev,uleft,y*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(urightDev,uright,y*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vupDev,vup,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vdownDev,vdown,x*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vleftDev,vleft,y*sizeof(FLDBL),hipMemcpyHostToDevice);
	hipMemcpy(vrightDev,vright,y*sizeof(FLDBL),hipMemcpyHostToDevice);
		hipfftDoubleComplex *data;
		hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*x*y);
	for (int ti=0;ti<t;ti++)
	{
		if (!(ti%10))
			printf("%d from %d\n",ti/10,t/10);
		hipfftHandle plan;

		streamingKernel<<<blocks,threads>>>(utempDev,vtempDev,unDev,vnDev,uupDev,udownDev,uleftDev, urightDev,vupDev,vdownDev,vleftDev, vrightDev,dx,dy,dt,x,y,nu);
		//printf("Start: %s\n",cudaGetErrorString(cudaGetLastError()));
		lineKernel<<<blocks,threads>>>(utempDev,vtempDev,fDev,x,y,dx,dy);
		//printf("After drift: %s\n",cudaGetErrorString(cudaGetLastError()));
		hipfftPlan2d(&plan, y, x, HIPFFT_D2Z);
		//printf("Making a plan: %s\n",cudaGetErrorString(cudaGetLastError()));
		hipfftExecD2Z(plan, fDev, data);
		//printf("Exec: %s\n",cudaGetErrorString(cudaGetLastError()));
		hipfftDestroy(plan);
		//printf("Destroying: %s\n",cudaGetErrorString(cudaGetLastError()));
		fourieKernel<<<sblocks,threads>>>(data, dx,dy,x,y);
		//printf("Fourie: %s\n",cudaGetErrorString(cudaGetLastError()));
		hipfftPlan2d(&plan, y, x, HIPFFT_Z2D);
		//printf("Making a plan: %s\n",cudaGetErrorString(cudaGetLastError()));
		hipfftExecZ2D(plan,data,pDev);
		//printf("Exec: %s\n",cudaGetErrorString(cudaGetLastError()));
		hipfftDestroy(plan);
		//printf("Destroying: %s\n",cudaGetErrorString(cudaGetLastError()));
		hipMemcpy(p,pDev,numBytes,hipMemcpyDeviceToHost);
		for (i=0;i<x;i++)
			for (j=0;j<y;j++)
			{
				p[j+x*i]=p[j+y*i]/x/y;
			}
		FLDBL pom=0;
		for (i=0;i<x/2;i++)
			for (j=0;j<y/2;j++)
			{
				pom=p[i+x*j];
				p[i+x*j]=p[j+y*i];
				p[j+y*i]=pom;
			}
	FILE *pressout = fopen("press.dat", "w");
	for(int i = 0; i < x; i++)
	{
		for(int j = 0; j < y; j++)
		{
			fprintf(pressout, "%lf\t%lf\t%lf\n", (double)i, (double)j, p[i+x*j]);
		}
		fprintf(pressout, "\n");
	}
	fclose(pressout);

		hipMemcpy(pDev,p,numBytes,hipMemcpyHostToDevice);
		//printf("Norm: %s\n",cudaGetErrorString(cudaGetLastError()));
		summaryKernel<<<lsblocks, threads>>>(pDev,utempDev,vtempDev,unDev,vnDev,uupDev,udownDev,uleftDev, urightDev,vupDev,vdownDev,vleftDev, vrightDev,x,y,dx,dy);
		//printf("Last: %s\n",cudaGetErrorString(cudaGetLastError()));

	//FILE *rout = fopen("r.dat", "w");
	//for(int i = 0; i < x; i++)
	//{
	//	for(int j = 0; j < y; j++)
	//	{
	//		fprintf(rout, "%lf\t%lf\t%lf\n", (double)i, (double)j, data[i+x*j].x);
	//	}
	//	fprintf(rout, "\n");
	//}
	//fclose(rout);


	}
		hipFree(data);
	hipMemcpy(un,unDev,numBytes,hipMemcpyDeviceToHost);
	hipMemcpy(vn,vnDev,numBytes,hipMemcpyDeviceToHost);
	hipMemcpy(uup,uupDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(udown,udownDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(uleft,uleftDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(uright,urightDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vup,vupDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vdown,vdownDev,x*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vleft,vleftDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipMemcpy(vright,vrightDev,y*sizeof(FLDBL),hipMemcpyDeviceToHost);
	hipFree(unDev);
	hipFree(vnDev);
	hipFree(uupDev);
	hipFree(udownDev);
	hipFree(uleftDev);
	hipFree(urightDev);
	hipFree(vupDev);
	hipFree(vdownDev);
	hipFree(vleftDev);
	hipFree(vrightDev);
	hipFree(utempDev);
	hipFree(vtempDev);
	hipFree(fDev);
	field_div(un,vn,x,y,dx,dy);


	for (i=1;i<x+1;i++)
		for(j=1;j<y+1;j++)
		{
			u[i+(x+2)*j]=un[i-1+x*(j-1)];
			v[i+(x+2)*j]=vn[i-1+x*(j-1)];
		}
	for(i=1;i<x+1;i++)
	{
		u[i]=uup[i-1];
		u[i+(x+2)*(y+1)]=udown[i-1];
		v[i]=vup[i-1];
		v[i+(x+2)*(y+1)]=vdown[i-1];
	}
	for(j=1;j<y+1;j++)
	{
		u[(x+2)*j]=uleft[j-1];
		u[x+1+(x+2)*j]=uright[j-1];
		v[(x+2)*j]=vleft[j-1];
		v[x+1+(x+2)*j]=vright[j-1];
	}
	delete [] un;
	delete [] vn;
	delete [] uleft;
	delete [] uright;
	delete [] uup;
	delete [] udown;
	delete [] vleft;
	delete [] vright;
	delete [] vup;
	delete [] vdown;
	delete [] f;
	delete [] p;
}
void field_div(FLDBL * un, FLDBL * vn, int x, int y, FLDBL dx, FLDBL dy)
{
	FLDBL divu=0;
	int maxi=0;
	int maxj=0;
	int i=0;
	int j=0;
	divu=0;
	for (i=1;i<x-1;i++)
		for (j=1;j<y-1;j++)
			if(divu<fabs((un[i+1+x*j]-un[i-1+x*j])/dx/2+(vn[i+x*(j+1)]-vn[i+x*(j-1)])/dy/2))
			{
				divu=fabs((un[i+1+x*j]-un[i-1+x*j])/dx/2+(vn[i+x*(j+1)]-vn[i+x*(j-1)])/dy/2);
				maxi=i;
				maxj=j;
			}
	printf("~~~i=%d,j=%d~\n",maxi,maxj);
	printf("~~~~div = %f;\n",divu);
	FLDBL omega=0;
	for (i=0;i<x-1;i++)
		for (j=0;j<y-1;j++)
			if(omega<abs(-(un[i+x*(j+1)]-un[i+x*j])/dy+(vn[i+1+x*j]-vn[i+x*j])/dx))
				omega=abs(-(un[i+x*(j+1)]-un[i+x*j])/dy+(vn[i+1+x*j]-vn[i+x*j])/dx);
	printf("~~~~omega=%f\n",omega);

}